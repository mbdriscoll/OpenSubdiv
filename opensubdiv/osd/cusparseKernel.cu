
#include <hip/hip_runtime.h>
#include <stdio.h>

#define THREADS_PER_BLOCK 512

__global__ void
expand(int src_numthreads, int nve,
  int* dst_rows, int* dst_cols, float* dst_vals,
  int* src_rows, int* src_cols, float* src_vals)
{
    int thread = threadIdx.x + blockIdx.x * blockDim.x;
    if (thread >= src_numthreads)
        return;

    int r = thread / nve; // src_row
    int k = thread % nve; // replica number

    int i = src_rows[r];
    int stride = src_rows[r+1]-src_rows[r];
    int dst_base = i*nve + k*stride;
    int src_base = src_rows[r];
    dst_rows[r*nve + k] = dst_base;

    for(i = src_rows[r]; i < src_rows[r+1]; i++) {
	    int offset = i - src_base;
	    int col = src_cols[i];
	    float val = src_vals[i];
	    dst_cols[dst_base+offset] = col*nve + k;
	    dst_vals[dst_base+offset] = val;
    }
}

__global__ void
spmv(int m, int nnz, const int* M_rows, const int* M_cols, const float* M_vals, const float* V_in, float* V_out)
{
    int row = threadIdx.x + blockIdx.x * blockDim.x;
    if (row >= m)
        return;

    register float answer = 0.0;
    int lb = M_rows[row],
        ub = M_rows[row+1];

    for(int offset = lb; offset < ub; offset++)
	answer += M_vals[offset] * V_in[ M_cols[offset] ];

    V_out[row] = answer;
}

extern "C" {

#include <hipsparse.h>

void
OsdCusparseExpand(int src_numrows, int factor,
    int* dst_rows, int* dst_cols, float* dst_vals,
    int* src_rows, int* src_cols, float* src_vals)
{
    int blks = (src_numrows*factor + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    expand<<<blks,THREADS_PER_BLOCK>>>(src_numrows, factor,
            dst_rows, dst_cols, dst_vals,
            src_rows, src_cols, src_vals);
}

hipsparseStatus_t
my_cusparseScsrmv(hipsparseHandle_t handle, hipsparseOperation_t transA,
    int m, int n, int nnz, float* alpha,
    hipsparseMatDescr_t descrA,
    const float *csrValA,
    const int *csrRowPtrA, const int *csrColIndA,
    const float *x, float* beta,
    float *y ) {

    const int* M_rows = csrRowPtrA;
    const int* M_cols = csrColIndA;
    const float* M_vals = csrValA;
    const float* V_in = x;
    float* V_out = y;

    int blks = (m + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK;
    spmv<<<blks,THREADS_PER_BLOCK>>>(m, nnz, M_rows, M_cols, M_vals, V_in, V_out);

    return HIPSPARSE_STATUS_SUCCESS;
}

} /* extern C */
